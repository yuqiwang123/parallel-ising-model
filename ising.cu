#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <stdio.h>
#include <string>
#include <stdlib.h>
#include <fstream>
#include <chrono>

#define N 1000
#define THREADS 100
#define ITERS 50

__global__ void init(signed char* lattice, const float* __restrict__ randvals) {
  long long tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
  if(tid >= N * N/2) return;
  signed char spin = (randvals[tid] < 0.5f) ? -1 : 1;
  lattice[tid] = spin;
}

__global__ void update(signed char* lattice, bool is_black, 
                       const signed char* __restrict__ op_lattice,
                       const float* __restrict__ randvals) {
  long long tid = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
  int i = tid / (N/2), j = tid % (N/2);
  if(i >= N || j >= N/2) return;
  int down = (i + 1 >= N) ? 0 : i + 1;
  int up = (i - 1 < 0) ? N - 1 : i - 1;
  int right = (j + 1 >= N/2) ? 0 : j + 1;
  int left = (j - 1 < 0) ? N/2 - 1 : j - 1;

  int joff;
  if (is_black) {
    joff = (i % 2) ? right : left;
  } else {
    joff = (i % 2) ? left : right;
  }

  signed char sum = op_lattice[down * N/2 + j] + op_lattice[i * N/2 + j] + op_lattice[up * N/2 + j] + op_lattice[i * N/2 + joff];
  signed char spin = lattice[i * N/2 + j];
  float T = 0.1f*2.26918531421f;
  float P = exp(-2.0f * (1.0/T) * sum * spin);
  if (randvals[i*N/2 + j] < P) {
    lattice[i*N/2 + j] = -spin;
  }
}

void output(signed char *lattice_b, signed char *lattice_w) {
  signed char *lattice, *black, *white;
  lattice = (signed char*)malloc(N*N * sizeof(*lattice));
  black = (signed char*)malloc(N*N/2 * sizeof(*black));
  white = (signed char*)malloc(N*N/2 * sizeof(*white));

  hipMemcpy(black, lattice_b, N*N/2 * sizeof(*lattice_b), hipMemcpyDeviceToHost);
  hipMemcpy(white, lattice_w, N*N/2 * sizeof(*lattice_w), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N/2; j++) {
      if (i % 2) {
        lattice[i*N+2*j+1] = black[i*N/2+j];
        lattice[i*N+2*j] = white[i*N/2+j];
      } else {
        lattice[i*N+2*j] = black[i*N/2+j];
        lattice[i*N+2*j+1] = white[i*N/2+j];
      }
    }
  }

  FILE* f = fopen("lattice.txt", "w");
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
       fprintf(f, "%i ", (int)lattice[i*N+j]);
       //f << (int)lattice[i*N+j] << " ";
    }
    fprintf(f, "\n");
  }
  fclose(f);

  free(lattice);
  free(black);
  free(white);
}

int main() {
  signed char *lattice_b, *lattice_w;
  hipMalloc(&lattice_b, N * N/2 * sizeof(*lattice_b));
  hipMalloc(&lattice_w, N * N/2 * sizeof(*lattice_w));

  hiprandGenerator_t crd;
  unsigned long long seed = 1111ULL;
  hiprandCreateGenerator(&crd, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
  hiprandSetPseudoRandomGeneratorSeed(crd, seed);
  float *randvals;
  hipMalloc(&randvals, N*N/2 * sizeof(*randvals));

  int blocks = (N*N/2 + THREADS - 1)/THREADS;
  hiprandGenerateUniform(crd, randvals, N*N/2);
  init<<<blocks, THREADS>>>(lattice_b, randvals);
  hiprandGenerateUniform(crd, randvals, N*N/2);
  init<<<blocks, THREADS>>>(lattice_w, randvals); 
  
  auto start = std::chrono::high_resolution_clock::now();
  for(int i = 0; i < ITERS; i++) {
    hiprandGenerateUniform(crd, randvals, N*N/2);
    update<<<blocks, THREADS>>>(lattice_b, true, lattice_w, randvals);
    hiprandGenerateUniform(crd, randvals, N*N/2);
    update<<<blocks, THREADS>>>(lattice_w, false, lattice_b, randvals);    
  }
  hipDeviceSynchronize();
  auto end = std::chrono::high_resolution_clock::now();
  double time = (double) std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  printf("updated time: %fs\n", time * 1e-6);

  //output
  output(lattice_b, lattice_w);
  return 0;	
}
